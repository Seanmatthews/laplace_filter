#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <array>
#include <assert.h>
#include <iostream>

#define checkCudaErrors(val) assert(hipSuccess == val)

const int NUM_CHANNELS = 3;
const int KERNEL_DIM = 5;
const int KERNEL_RADIUS = (KERNEL_DIM - 1) / 2;
__device__ __constant__ float d_KERNEL[KERNEL_DIM];
texture<uint8_t, hipTextureType2D> TEXTURE8U;


// Loop unrolling templates for maximixing instruction-level parallelism
template<int i>//, bool row>
__device__ float convolutionUnroll(const float x, const float y)
{
    float xValRow = x + (float)NUM_CHANNELS * (float)(KERNEL_RADIUS - i);
    float yValCol = y + (float)KERNEL_RADIUS - (float)i;
    
    return (float)tex2D(TEXTURE8U, xValRow, y) * d_KERNEL[i] +
        (float)tex2D(TEXTURE8U, x, yValCol) * d_KERNEL[i] +
        convolutionUnroll<i-1>(x,y);
}

// Loop unrolling base case template specialization
template<>
__device__ float convolutionUnroll<-1>(const float x, const float y)
{
    return 0;
}


/*
template<int i>
__device__ float convolutionUnrollCol(const float x, const float y)
{
    const int iRow = 2 * KERNEL_RADIUS;
    float yVal = y + (float)KERNEL_RADIUS - (float)i;
    return convolutionUnrollCol<i-1>(x, y) +
        convolutionUnrollRow<iRow>(x, yVal);
}

template<>
__device__ float convolutionUnrollCol<-1>(const float x, const float y)
{
    return 0;
}
*/



// Texture convolution
__global__ void convolution(uint8_t* dst, const int imgWidth,
                            const int imgHeight)
{
    const int xIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const int yIdx = threadIdx.y + blockIdx.y * blockDim.y;
    const float x = (float)xIdx + 0.5f;
    const float y = (float)yIdx + 0.5f;
    const int i = 2 * KERNEL_RADIUS;
    
    if (xIdx < imgWidth && yIdx < imgHeight)
    {
        dst[yIdx * imgWidth + xIdx] = 
            (uint8_t)roundf(convolutionUnroll<i>(x, y));
//            (uint8_t)roundf(convolutionUnrollCol<i>(x, y));
    }
}


cv::Mat createLOGKernel1D(int ksize, float sigma)
{
    using namespace cv;

    float std2 = sigma * sigma;
    std::vector<float> seq(ksize);
    std::iota(begin(seq), end(seq), -(ksize-1)/2);
    Mat_<float> kSeq(1, ksize, seq.data());

    Mat XX;
    multiply(kSeq, kSeq, XX);

    Mat H = -(XX / (2.*std2));
    exp(H, H);

    double minVal, maxVal;
    minMaxIdx(H, &minVal, &maxVal);
    Mat mask = H < std::numeric_limits<float>::epsilon()*maxVal;
    H.setTo(0, mask);

    float sumh = sum(H)[0];
    if (sumh != 0) H /= sumh;

    Mat H1;
    multiply(H, (XX - 2.*std2)/(std2 * std2), H1);
    H = H1 - sum(H1)[0] / ksize;
    return H;    
}


// Calculates Laplace of Gaussian kernel
cv::Mat createLOGKernel2D(int ksize, double sigma)
{
    using namespace cv;
    using namespace std;

    Mat kernel(ksize, ksize, CV_64F);
    Mat X(ksize, ksize, CV_64F);
    Mat Y(ksize, ksize, CV_64F);
    int siz = (ksize-1)/2;
    double std2 = sigma*sigma;

    vector<double> seq(ksize);
    iota(begin(seq), end(seq), -siz);
    Mat seqX(1, ksize, CV_64F, seq.data());
    Mat seqY(ksize, 1, CV_64F, seq.data());
    repeat(seqX, ksize, 1, X);
    repeat(seqY, 1, ksize, Y);

    Mat XX, YY;
    multiply(X, X, XX);
    multiply(Y, Y, YY);

    Mat H = -(XX + YY) / (2.*std2);
    exp(H, H);

    double minVal, maxVal;
    minMaxIdx(H, &minVal, &maxVal);
    Mat mask = H < numeric_limits<double>::epsilon()*maxVal;
    H.setTo(0, mask);

    double sumh = sum(H)[0];
    if (sumh != 0) H /= sumh;

    Mat H1;
    multiply(H, (XX + YY - 2*std2) / (std2*std2), H1);
    H = H1 - sum(H1)[0] / (ksize*ksize);
    return H;
}


// A Laplacian morphological operation boils down to the convolution
// of a Laplacian-- Laplacian of Gaussian (LoG) in this case kernel over
// the image data. Separable texture-based convolution is a quick way
// to perform convolutions in CUDA, but Laplacian kernels are not
// separable. To get around this, I perform separate vertical and
// horizontal convolutions with a 1D LoG kernel and its transpose
// and summing the results.
int main(int argc, char** argv)
{
    using namespace cv;
    using namespace std;

    if (argc < 4)
    {
        cout << "Usage: laplacefilter <image path> <image width> <image height> [gaussian sigma]\n";
        return -1;
    }

    bool saveImages = true;
    float sigma = 0.5;

    if (argc >= 5) sigma = atof(argv[4]);
    
    // Load image
    int width = atoi(argv[2]);
    int height = atoi(argv[3]);
    int numBytes = 6 * (width*height)/4;
    unsigned char bytes[numBytes];
    ifstream in(argv[1], ifstream::binary|ifstream::in);
    in.read((char*)bytes, numBytes);
    Mat yuv(height+height/2, width, CV_8UC1, bytes);
    Mat img(height, width, CV_8UC3);
    cvtColor(yuv, img, COLOR_YUV420p2RGB);    
    int fullWidth = img.channels() * img.cols;
    
    if (img.empty())
    {
        cout << "Empty image!\n";
        return -1;
    }

    assert(img.rows == height);
    assert(img.cols == width);
    assert(img.channels() == NUM_CHANNELS);

    // Calculate grid and block sizes.
    int numImgBytes = img.step[0] * img.rows;
    dim3 threads(16, 12); // 6 * 32 (warp size)
    dim3 blocks(ceil((float)fullWidth/threads.x), ceil((float)img.rows/threads.y));
    
    cout << img.cols << " " << img.rows << " " << img.channels() << endl;
    cout << "num Bytes: " << numImgBytes << endl;
    cout << "grid: (" << blocks.x << ", " << blocks.y << ") block: (" << threads.x << ", " << threads.y << ")\n"; 

    Mat logKernel = createLOGKernel1D(KERNEL_DIM, sigma);
//    Mat logKernel = createLOGKernel2D(KERNEL_DIM, sigma);
    vector<uint8_t> h_result(numImgBytes);
    uint8_t *d_result;
    hipArray* cu_imgArray;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate array memory & bind
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&cu_imgArray, &channelDesc, fullWidth, img.rows));        
    checkCudaErrors(hipBindTextureToArray(TEXTURE8U, cu_imgArray));
    checkCudaErrors(hipMalloc((void**)&d_result, numImgBytes));    
    checkCudaErrors(hipMemset(d_result, 0, numImgBytes));
    
    // Copy everything to the device
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_KERNEL), logKernel.data, KERNEL_DIM * sizeof(float)));
    checkCudaErrors(hipMemcpyToArray(cu_imgArray, 0, 0, img.data, numImgBytes, hipMemcpyHostToDevice));

    // Run & time kernel
    hipEventRecord(start);
    convolution<<<blocks, threads>>>(d_result, fullWidth, img.rows);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Copy & free gpu mempory
    checkCudaErrors(hipMemcpy(h_result.data(), d_result, numImgBytes, hipMemcpyDeviceToHost));        
    checkCudaErrors(hipFree(d_result));
    checkCudaErrors(hipFreeArray(cu_imgArray));
    
    // Record run duration & throughput
    float time;
    hipEventElapsedTime(&time, start, stop);
    cout << "Kernel runtime: " << time << " ms\n";
    cout << "Bandwidth: " << numImgBytes*2/time/1e6 << " GB/s\n";
    cout << width * height * 1e-6 / (0.001 * time) << " MPixels/s\n";
    
    // Display & save
    Mat resultImg(img.rows, img.cols, img.type(), h_result.data());
    Mat bgr[img.channels()];
    split(resultImg, bgr);
    vector<string> channelNames = {"redChannel", "greenChannel", "blueChannel"}; 
    
    for (int i=0; i<img.channels(); ++i)
    {
        int idx = img.channels() - i - 1;
        namedWindow(channelNames[idx], WINDOW_NORMAL);
        imshow(channelNames[idx], bgr[i]);
        if (saveImages) imwrite(channelNames[idx] + ".jpg", bgr[i]);
    }

    if (saveImages) imwrite("allChannels.jpg", resultImg);
    namedWindow("All Channels", WINDOW_NORMAL);
    imshow("All Channels", resultImg);
    waitKey(0);
    
    return 0;
}

